#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016
 *
 * Cristián Maureira-Fredes <cmaureirafredes@gmail.com>
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright
 * notice, this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright
 * notice, this list of conditions and the following disclaimer in the
 * documentation and/or other materials provided with the distribution.
 *
 * 3. The name of the author may not be used to endorse or promote
 * products derived from this software without specific prior written
 * permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR ``AS IS'' AND ANY EXPRESS
 * OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE
 * GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER
 * IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
 * OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN
 * IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 */
#undef _GLIBCXX_ATOMIC_BUILTINS
#include "Hermite4GPU.cuh"
#include "nvToolsExt.h"

/** Constructor that uses its parent one.
 * Additionally handles the split of the particles of the system among the available
 * GPUs, allocation of the variables, and defining widely use sizes for arrays.
 */
Hermite4GPU::Hermite4GPU(NbodySystem *ns, Logger *logger, NbodyUtils *nu)
            : Hermite4(ns, logger, nu)
{
    smem = sizeof(Predictor) * BSIZE;
    smem_reduce = sizeof(Forces) * NJBLOCK + 1;

    int detected_gpus;
    CSC(hipGetDeviceCount(&detected_gpus));

    if (ns->gpus > 0)
    {
        gpus = ns->gpus;
    }
    else
    {
        gpus = detected_gpus;
    }

    if (detected_gpus > gpus)
    {
        std::string s = "";
        s += std::string("Not using all the available GPUs: ");
        s += std::string(SSTR(gpus));
        s += std::string(" of ");
        s += std::string(SSTR(detected_gpus));
        logger->log_warning(s);
    }

    logger->log_info(std::string("GPUs: ")+std::string(SSTR(gpus)));

    std::string ss = "";
    ss += std::string("Splitting ");
    ss += std::string(SSTR(ns->n));
    ss += std::string(" particles in ");
    ss += std::string(SSTR(gpus));
    ss += std::string(" GPUs");
    logger->log_info(ss);

    if (ns->n % gpus == 0)
    {
        size_t size = ns->n/gpus;
        for ( int g = 0; g < gpus; g++)
            n_part[g] = size;
    }
    else
    {
        size_t size = std::ceil(ns->n/(float)gpus);
        for ( int g = 0; g < gpus; g++)
        {
            if (ns->n - size*(g+1) > 0)
                n_part[g] = size;
            else
                n_part[g] = ns->n - size*g;
        }
    }

    for(int g = 0; g < gpus; g++)
    {
        std::string sss = "";
        sss += std::string("GPU ");
        sss += std::string(SSTR(g));
        sss += std::string(" particles: ");
        sss += std::string(SSTR(n_part[g]));
        logger->log_info(sss);
    }

    i1_size = ns->n * sizeof(int);
    d1_size = ns->n * sizeof(double);
    d3_size = ns->n * sizeof(double3);
    d4_size = ns->n * sizeof(double4);
    ff_size = ns->n * sizeof(Forces);
    pp_size = ns->n * sizeof(Predictor);

    alloc_arrays_device();
}

/** Destructor in charge of memory deallocation */
Hermite4GPU::~Hermite4GPU()
{
    free_arrays_device();
}

/** Method in charge of allocating the data structures on the available GPUs,
 * also initializing all the arrays to zero
 */
void Hermite4GPU::alloc_arrays_device()
{
    for(int g = 0; g < gpus; g++)
    {
        // Setting GPU
        CSC(hipSetDevice(g));

        CSC(hipMalloc((void**)&ns->d_r[g], d4_size));
        CSC(hipMalloc((void**)&ns->d_v[g], d4_size));
        CSC(hipMalloc((void**)&ns->d_f[g], ff_size));
        CSC(hipMalloc((void**)&ns->d_p[g], pp_size));
        CSC(hipMalloc((void**)&ns->d_t[g], d1_size));
        CSC(hipMalloc((void**)&ns->d_i[g], pp_size));
        CSC(hipMalloc((void**)&ns->d_dt[g], d1_size));

        CSC(hipMalloc((void**)&ns->d_a2[g], d3_size));
        CSC(hipMalloc((void**)&ns->d_a3[g], d3_size));
        CSC(hipMalloc((void**)&ns->d_old[g], ff_size));

        CSC(hipMalloc((void**)&ns->d_ekin[g], d1_size));
        CSC(hipMalloc((void**)&ns->d_epot[g], d1_size));
        CSC(hipMalloc((void**)&ns->d_move[g], i1_size));
        CSC(hipMalloc((void**)&ns->d_fout[g], ff_size * NJBLOCK));
        CSC(hipMalloc((void**)&ns->d_fout_tmp[g], ff_size * NJBLOCK));

        CSC(hipMemset(ns->d_r[g], 0, d4_size));
        CSC(hipMemset(ns->d_v[g], 0, d4_size));
        CSC(hipMemset(ns->d_f[g], 0, ff_size));
        CSC(hipMemset(ns->d_p[g], 0, pp_size));
        CSC(hipMemset(ns->d_t[g], 0, d1_size));
        CSC(hipMemset(ns->d_i[g], 0, pp_size));
        CSC(hipMemset(ns->d_dt[g], 0, d1_size));
        CSC(hipMemset(ns->d_ekin[g], 0, d1_size));
        CSC(hipMemset(ns->d_epot[g], 0, d1_size));
        CSC(hipMemset(ns->d_move[g], 0, i1_size));
        CSC(hipMemset(ns->d_fout[g], 0, ff_size * NJBLOCK));
        CSC(hipMemset(ns->d_fout_tmp[g], 0, ff_size * NJBLOCK));

        ns->h_fout_gpu[g] = new Forces[ns->n*NJBLOCK];
    }

    // Extra CPU array
    ns->h_fout_tmp = new Forces[ns->n*NJBLOCK];
}

/** Method in charge of deallocating the data structures on the available GPUs.
 */
void Hermite4GPU::free_arrays_device()
{

    for(int g = 0; g < gpus; g++)
    {
        // Setting GPU
        CSC(hipSetDevice(g));

        CSC(hipFree(ns->d_r[g]));
        CSC(hipFree(ns->d_v[g]));
        CSC(hipFree(ns->d_f[g]));
        CSC(hipFree(ns->d_p[g]));
        CSC(hipFree(ns->d_t[g]));
        CSC(hipFree(ns->d_i[g]));
        CSC(hipFree(ns->d_dt[g]));

        CSC(hipFree(ns->d_a2[g]));
        CSC(hipFree(ns->d_a3[g]));
        CSC(hipFree(ns->d_old[g]));

        CSC(hipFree(ns->d_ekin[g]));
        CSC(hipFree(ns->d_epot[g]));
        CSC(hipFree(ns->d_move[g]));
        CSC(hipFree(ns->d_fout[g]));
        CSC(hipFree(ns->d_fout_tmp[g]));
        delete ns->h_fout_gpu[g];
    }

    delete ns->h_fout_tmp;
    //delete ns->h_fout_gpu;
}

/** Method in charge of the prediction step.
 * This can be use on the CPU (commented section) or on the GPUs.
 * The reason of having both reasons, is the improvement is not much for small
 * amount of particles.
 */
void Hermite4GPU::predicted_pos_vel(double ITIME)
{
    ns->gtime.prediction_ini = omp_get_wtime();
    //#pragma omp parallel for
    //for (int i = 0; i < ns->n; i++)
    //{
    //    double dt  = ITIME - ns->h_t[i];
    //    double dt2 = 0.5*(dt  * dt);
    //    double dt3 = 0.166666666666666*(dt * dt * dt);

    //    Forces ff = ns->h_f[i];
    //    double4 rr = ns->h_r[i];
    //    double4 vv = ns->h_v[i];

    //    ns->h_p[i].r[0] = (dt3 * ff.a1[0]) + (dt2 * ff.a[0]) + (dt * vv.x) + rr.x;
    //    ns->h_p[i].r[1] = (dt3 * ff.a1[1]) + (dt2 * ff.a[1]) + (dt * vv.y) + rr.y;
    //    ns->h_p[i].r[2] = (dt3 * ff.a1[2]) + (dt2 * ff.a[2]) + (dt * vv.z) + rr.z;

    //    ns->h_p[i].v[0] = (dt2 * ff.a1[0]) + (dt * ff.a[0]) + vv.x;
    //    ns->h_p[i].v[1] = (dt2 * ff.a1[1]) + (dt * ff.a[1]) + vv.y;
    //    ns->h_p[i].v[2] = (dt2 * ff.a1[2]) + (dt * ff.a[2]) + vv.z;

    //    ns->h_p[i].m = rr.w;
    //}

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));
        int shift = g*n_part[g-1];
        size_t ff_size = n_part[g] * sizeof(Forces);
        size_t d4_size = n_part[g] * sizeof(double4);
        size_t d1_size = n_part[g] * sizeof(double);

        CSC(hipMemcpyAsync(ns->d_f[g], ns->h_f + shift, ff_size, hipMemcpyHostToDevice, 0));
        CSC(hipMemcpyAsync(ns->d_r[g], ns->h_r + shift, d4_size, hipMemcpyHostToDevice, 0));
        CSC(hipMemcpyAsync(ns->d_v[g], ns->h_v + shift, d4_size, hipMemcpyHostToDevice, 0));
        CSC(hipMemcpyAsync(ns->d_t[g], ns->h_t + shift, d1_size, hipMemcpyHostToDevice, 0));
    }

    // Executing kernels
    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        nthreads = BSIZE;
        nblocks = std::ceil(n_part[g]/(float)nthreads);

        k_prediction <<< nblocks, nthreads >>> (ns->d_f[g],
                                                ns->d_r[g],
                                                ns->d_v[g],
                                                ns->d_t[g],
                                                ns->d_p[g],
                                                n_part[g],
                                                ITIME);
        get_kernel_error();
    }

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));
        size_t slice = g*n_part[g-1];
        size_t pp_size = n_part[g] * sizeof(Predictor);

        CSC(hipMemcpyAsync(&ns->h_p[slice], ns->d_p[g], pp_size, hipMemcpyDeviceToHost, 0));
    }

    ns->gtime.prediction_end += omp_get_wtime() - ns->gtime.prediction_ini;
}

/** Method in charge of the corrector step.
 * This is not implemented on the GPU because the benefit was not much
 * for small amount of particles.
 */
void Hermite4GPU::correction_pos_vel(double ITIME, unsigned int nact)
{
    // Timer
    ns->gtime.correction_ini = omp_get_wtime();

    for (int g = 0; g < gpus; g++) {
      CSC(hipSetDevice(g));
      int shift = g*n_part[g-1];
      size_t ff_size = n_part[g] * sizeof(Forces);
      size_t d4_size = n_part[g] * sizeof(double4);
      size_t d3_size = n_part[g] * sizeof(double3);
      size_t d1_size = n_part[g] * sizeof(double);
      // nact is only correct if 1 gpu (which we are doing). if more, then need to dynamically make a new n_part[g]
      size_t  i_size = nact * sizeof(unsigned int);

      CSC(hipMemcpyAsync(ns->d_f[g], ns->h_f + shift, ff_size, hipMemcpyHostToDevice, 0));
      CSC(hipMemcpyAsync(ns->d_old[g], ns->h_old + shift, ff_size, hipMemcpyHostToDevice, 0));
      // CSC(hipMemcpyAsync(ns->d_r[g], ns->h_r + shift, d4_size, hipMemcpyHostToDevice, 0));
      // CSC(hipMemcpyAsync(ns->d_v[g], ns->h_v + shift, d4_size, hipMemcpyHostToDevice, 0));
      // CSC(hipMemcpyAsync(ns->d_t[g], ns->h_t + shift, d1_size, hipMemcpyHostToDevice, 0));
      CSC(hipMemcpyAsync(ns->d_dt[g], ns->h_dt + shift, d1_size, hipMemcpyHostToDevice, 0));
      CSC(hipMemcpyAsync(ns->d_move[g], ns->h_move + shift, i_size, hipMemcpyHostToDevice, 0));
    }

    // Executing kernels
    for (int g = 0; g < gpus; g++)
    {
      CSC(hipSetDevice(g));

      nthreads = BSIZE;
      // nblocks = std::ceil(n_part[g]/(float)nthreads);
      nblocks = std::ceil(nact/(float)nthreads); // nact, since this is only doing that many iterations

      k_correction <<< nblocks, nthreads >>> (ns->d_move[g],
                                              ns->d_f[g],
                                              ns->d_old[g],
                                              ns->d_p[g],
                                              ns->d_r[g],
                                              ns->d_v[g],
                                              ns->d_t[g],
                                              ns->d_dt[g],
                                              ns->d_a2[g],
                                              ns->d_a3[g],
                                              nact,
                                              ITIME,
                                              ns->eta);
      get_kernel_error();
    }

    for(int g = 0; g < gpus; g++)
    {
        // t, dt, a2, a3
        CSC(hipSetDevice(g));
        size_t slice = g*n_part[g-1];
        size_t d1_size = n_part[g] * sizeof(double);
        size_t d4_size = n_part[g] * sizeof(double4);

        CSC(hipMemcpyAsync(&ns->h_t[slice], ns->d_t[g], d1_size, hipMemcpyDeviceToHost, 0));
        CSC(hipMemcpyAsync(&ns->h_dt[slice], ns->d_dt[g], d1_size, hipMemcpyDeviceToHost, 0));
        CSC(hipMemcpyAsync(&ns->h_r[slice], ns->d_r[g], d4_size, hipMemcpyDeviceToHost, 0));
        CSC(hipMemcpyAsync(&ns->h_v[slice], ns->d_v[g], d4_size, hipMemcpyDeviceToHost, 0));
    }

    ns->gtime.correction_end += omp_get_wtime() - ns->gtime.correction_ini;
}

/** Method in charge of the initialization of all the particle's acceleration
 * and first derivative of the system, at the begining of the simulation.
 */
void Hermite4GPU::init_acc_jrk()
{
    size_t pp_size = ns->n * sizeof(Predictor);

    // Copying arrays to device
    #pragma omp parallel for num_threads(gpus)
    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        // All this information from the predictors is needed by each device
        CSC(hipMemcpy(ns->d_p[g], ns->h_p, pp_size, hipMemcpyHostToDevice));
        //CSC(hipMemcpyAsync(ns->d_p[g], ns->h_p, pp_size, hipMemcpyHostToDevice, 0));
    }

    // Executing kernels
    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        nthreads = BSIZE;
        nblocks = std::ceil(n_part[g]/(float)nthreads);

        k_init_acc_jrk <<< nblocks, nthreads, smem >>> (ns->d_p[g],
                                                        ns->d_f[g],
                                                        ns->n,
                                                        ns->e2,
                                                        g,
                                                        n_part[g]);
        get_kernel_error();
    }

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        size_t chunk = n_part[g]*sizeof(Forces);
        size_t slice = g*n_part[g-1];

        CSC(hipMemcpy(&ns->h_f[slice], ns->d_f[g], chunk, hipMemcpyDeviceToHost));
        //CSC(hipMemcpyAsync(&ns->h_f[slice], ns->d_f[g], chunk, hipMemcpyDeviceToHost, 0));
    }
}

/** Method in charge of the force interaction between \f$N_{act}\f$ and the whole
 * system.
 *  First there is a tmp construction of predictors to be send to the GPUs.
 *  Then the data is copied to the devices.
 *  The first kernel perform the preliminary calculation of the forces in JPBLOCKS.
 *  The second kernel, reduction, is in charge of summing all the preliminary forces
 *  to the final value for all the active particles.
 */
void Hermite4GPU::update_acc_jrk(unsigned int nact)
{
    // Timer begin
    ns->gtime.update_ini = omp_get_wtime();

    //for(int g = 0; g < gpus; g++)
    //{
    //    if (n_part[g] > 0)
    //    {
    //        size_t pp_size = n_part[g] * sizeof(Predictor);
    //        int shift = g*n_part[g-1];

    //        CSC(hipSetDevice(g));
    //        // Copying to the device the predicted r and v
    //        //CSC(hipMemcpy(ns->d_p[g], ns->h_p + shift, pp_size, hipMemcpyHostToDevice));
    //        CSC(hipMemcpyAsync(ns->d_p[g], ns->h_p + shift, pp_size, hipMemcpyHostToDevice, 0));
    //    }
    //}

    // Fill the h_i Predictor array with the particles that we need to move
    // // nvtxRangePushA("nact for loop");
    //
    // #pragma omp parallel for
    // for (int i = 0; i < nact; i++)
    // {
    //     ns->h_i[i] = ns->h_p[ns->h_move[i]];
    // }

    char nacts[128];
    sprintf(nacts, "nact for loop %d", nact);
    nvtxRangePushA(nacts);
    for(int g = 0; g < gpus; g++)
    {
        if (n_part[g] > 0)
        {
            CSC(hipSetDevice(g));
            // Copy to the GPU (d_i) the preddictor host array (h_i)
            size_t chunk = nact * sizeof(unsigned int);
            // CSC(hipMemcpyAsync(ns->d_i[g], ns->h_i, chunk, hipMemcpyHostToDevice, 0));
            CSC(hipMemcpyAsync(ns->d_move[g], ns->h_move, chunk, hipMemcpyHostToDevice, 0));
        }
    }
    nvtxRangePop();

    ns->gtime.grav_ini = omp_get_wtime();
    for(int g = 0; g < gpus; g++)
    {
        if (n_part[g] > 0)
        {
            CSC(hipSetDevice(g));
            // Blocks, threads and shared memory configuration
            int  nact_blocks = 1 + (nact-1)/BSIZE;
            dim3 nblocks(nact_blocks, NJBLOCK, 1);
            dim3 nthreads(BSIZE, 1, 1);

            // Kernel to update the forces for the particles in d_i
            k_update <<< nblocks, nthreads, smem >>> (ns->d_move[g],
                                                      ns->d_i[g],
                                                      ns->d_p[g], // partial
                                                      ns->d_fout[g],
                                                      n_part[g], // former N
                                                      nact,
                                                      ns->e2);
        }
    }

    ns->gtime.grav_end += omp_get_wtime() - ns->gtime.grav_ini;

    ns->gtime.reduce_ini = omp_get_wtime();
    for(int g = 0; g < gpus; g++)
    {
        size_t chunk = 2<<14;
        if (n_part[g] > 0)
        {
            CSC(hipSetDevice(g));
            // Blocks, threads and shared memory configuration for the reduction.
            if (nact <= chunk) // limit 32768
            {
                dim3 rgrid   (nact,   1, 1);
                dim3 rthreads(NJBLOCK, 1, 1);

                // Kernel to reduce que temp array with the forces
                k_reduce <<< rgrid, rthreads, smem_reduce >>>(ns->d_fout[g],
                                                            ns->d_fout_tmp[g],
                                                            0,
                                                            0);
            }
            else
            {

                int smax = std::ceil(nact/(float)chunk);
                unsigned int shift = 0;
                size_t size_launch = 0;

                for(unsigned int s = 0; shift < nact; s++)
                {
                    // shift_id : s
                    // shift: moving pointer
                    // size_launch: chunk to multipy by Forces size
                    if (nact < shift + chunk)
                        size_launch = nact-shift;
                    else
                        size_launch = chunk;

                    dim3 rgrid   (size_launch,   1, 1);
                    dim3 rthreads(NJBLOCK, 1, 1);
                    k_reduce <<< rgrid, rthreads, smem_reduce >>>(ns->d_fout[g],
                                                                  ns->d_fout_tmp[g]+shift,
                                                                  s,
                                                                  shift);


                    shift += chunk;
                }
            }
        }
    }
    ns->gtime.reduce_end += omp_get_wtime() - ns->gtime.reduce_ini;

    // Update forces in the host
    ns->gtime.reduce_forces_ini = omp_get_wtime();

    int g = 0; // g can only be 0 here. Only use 1 GPU for this.
    if (gpus > 1) {
      char gpu_msg[128];
      sprintf(gpu_msg, "GPUS (%d) GTR THAN 1!!!!!", gpus);
      logger->log(1, gpu_msg);
    }
    CSC(hipSetDevice(g));

    nthreads = BSIZE;
    nblocks = std::ceil(nact/(float)nthreads);

    k_assign_forces <<< nblocks, nthreads >>> (ns->d_move[g],
                                               ns->d_fout_tmp[g],
                                               ns->d_f[g],
                                               nact);
    get_kernel_error();


    ///// CPU reduce forces across GPUs. Updates big F array with small subset F array
    ///// The zero initialization is because it assumes multiple GPUs and so must add.
    //// We will assume 1 GPU for now.
    // #pragma omp parallel for
    // for (int i = 0; i < nact; i++)
    // {
    //     int id = ns->h_move[i];
    //     ns->h_f[id].a[0] = 0.0;
    //     ns->h_f[id].a[1] = 0.0;
    //     ns->h_f[id].a[2] = 0.0;
    //     ns->h_f[id].a1[0] = 0.0;
    //     ns->h_f[id].a1[1] = 0.0;
    //     ns->h_f[id].a1[2] = 0.0;
    //
    //     for(int g = 0; g < gpus; g++)
    //     {
    //         if (n_part[g] > 0)
    //         {
    //             ns->h_f[id] += ns->h_fout_gpu[g][i];
    //         }
    //     }
    // }

    for(int g = 0; g < gpus; g++)
    {
        if (n_part[g] > 0)
        {
            CSC(hipSetDevice(g));

            size_t slice = g*n_part[g-1];
            size_t ff_size = n_part[g] * sizeof(Forces);

            // Copy from the GPU all forces, which have been updated
            // old copy:
            //CSC(hipMemcpy(ns->h_fout_gpu[g], ns->d_fout_tmp[g], chunk, hipMemcpyDeviceToHost));
            CSC(hipMemcpyAsync(&ns->h_f[slice], ns->d_f[g], ff_size, hipMemcpyDeviceToHost, 0));
        }
    }


    ns->gtime.reduce_forces_end += omp_get_wtime() - ns->gtime.reduce_forces_ini;

    // Timer end
    ns->gtime.update_end += (omp_get_wtime() - ns->gtime.update_ini);
}

/** Method in charge of calculating the potential and kinetic energy
 * on the GPU devices
 */
double Hermite4GPU::get_energy_gpu()
{
    double time_energy_ini = omp_get_wtime();

    size_t d4_size = ns->n * sizeof(double4);

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        CSC(hipMemcpyAsync(ns->d_r[g], ns->h_r, d4_size, hipMemcpyHostToDevice, 0));
        CSC(hipMemcpyAsync(ns->d_v[g], ns->h_v, d4_size, hipMemcpyHostToDevice, 0));
    }

    int nthreads = BSIZE;
    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        int nblocks = std::ceil(n_part[g]/(float)nthreads);
        k_energy <<< nblocks, nthreads >>> (ns->d_r[g],
                                            ns->d_v[g],
                                            ns->d_ekin[g],
                                            ns->d_epot[g],
                                            ns->n,
                                            n_part[g],
                                            g);
    }

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        size_t chunk = n_part[g]*sizeof(double);
        size_t slice = g*n_part[g-1];

        CSC(hipMemcpyAsync(&ns->h_ekin[slice], ns->d_ekin[g], chunk, hipMemcpyDeviceToHost, 0));
        CSC(hipMemcpyAsync(&ns->h_epot[slice], ns->d_epot[g], chunk, hipMemcpyDeviceToHost, 0));
    }

    // Reduction on CPU
    ns->en.kinetic = 0.0;
    ns->en.potential = 0.0;
    for (int i = 0; i < ns->n; i++)
    {
        ns->en.kinetic   += ns->h_ekin[i];
        ns->en.potential += ns->h_epot[i];
    }

    double time_energy_end = omp_get_wtime() - time_energy_ini;

    return ns->en.kinetic + ns->en.potential;
}

/** Method that get the last kernel error if the code is running with the DEBUG
 * flag
 */
void Hermite4GPU::get_kernel_error()
{
    #ifdef KERNEL_ERROR_DEBUG
    logger->log_error(std::string(hipGetErrorString(hipGetLastError())));
    #endif
}

/** Method to start the device timer
 */
void Hermite4GPU::gpu_timer_start(){
    hipEventRecord(start);
}

/** Method that ends the device timer
 */
float Hermite4GPU::gpu_timer_stop(std::string f){
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float msec = 0;
    hipEventElapsedTime(&msec, start, stop);
    #if KERNEL_TIME
    if (f != "")
    {
        std::string s = "";
        s += std::string("Kernel ");
        s += std::string(SSTR(f));
        s += std::string(" : ");
        s += std::string(SSTR(msec));
        logger->log_info(s)
    }
    #endif
    return msec;
}

/** This method is not implemented becasue we use a CUDA kernel
 * to perfom the force calculation, not a host method.
 */
void Hermite4GPU::force_calculation(const Predictor &pi, const Predictor &pj, Forces &fi) {}



/**
Some temporary functions that should be moved to GPU later
**/
/**
Workshopping below here

**/
/** Vector magnitude calculation */
double Hermite4GPU::get_magnitude(double x, double y, double z)
{
    return sqrt(x*x + y*y + z*z);
}

/** Time step calculation */
double Hermite4GPU::get_timestep_normal(unsigned int i, float ETA)
{
    // Calculating a_{1,i}^{(2)} = a_{0,i}^{(2)} + dt * a_{0,i}^{(3)}
    double ax1_2 = ns->h_a2[i].x + ns->h_dt[i] * ns->h_a3[i].x;
    double ay1_2 = ns->h_a2[i].y + ns->h_dt[i] * ns->h_a3[i].y;
    double az1_2 = ns->h_a2[i].z + ns->h_dt[i] * ns->h_a3[i].z;

    // |a_{1,i}|
    double abs_a1 = get_magnitude(ns->h_f[i].a[0],
                                  ns->h_f[i].a[1],
                                  ns->h_f[i].a[2]);
    // |j_{1,i}|
    double abs_j1 = get_magnitude(ns->h_f[i].a1[0],
                                  ns->h_f[i].a1[1],
                                  ns->h_f[i].a1[2]);
    // |j_{1,i}|^{2}
    double abs_j12  = abs_j1 * abs_j1;
    // a_{1,i}^{(3)} = a_{0,i}^{(3)} because the 3rd-order interpolation
    double abs_a1_3 = get_magnitude(ns->h_a3[i].x,
                                    ns->h_a3[i].y,
                                    ns->h_a3[i].z);
    // |a_{1,i}^{(2)}|
    double abs_a1_2 = get_magnitude(ax1_2, ay1_2, az1_2);
    // |a_{1,i}^{(2)}|^{2}
    double abs_a1_22  = abs_a1_2 * abs_a1_2;

    double normal_dt = sqrt(ETA * ((abs_a1 * abs_a1_2 + abs_j12) / (abs_j1 * abs_a1_3 + abs_a1_22)));
    return normal_dt;
}

/** Normalization of the timestep.
 * This method take care of the limits conditions to avoid large jumps between
 * the timestep distribution
 */
double Hermite4GPU::normalize_dt(double new_dt, double old_dt, double t, unsigned int i)
{
    if (new_dt <= old_dt/8)
    {
        new_dt = D_TIME_MIN;
    }
    else if ( old_dt/8 < new_dt && new_dt <= old_dt/4)
    {
        new_dt = old_dt / 8;
    }
    else if ( old_dt/4 < new_dt && new_dt <= old_dt/2)
    {
        new_dt = old_dt / 4;
    }
    else if ( old_dt/2 < new_dt && new_dt <= old_dt)
    {
        new_dt = old_dt / 2;
    }
    else if ( old_dt < new_dt && new_dt <= old_dt * 2)
    {
        new_dt = old_dt;
    }
    else if (2 * old_dt < new_dt)
    {
        double val = t/(2 * old_dt);
        //float val = t/(2 * old_dt);
        if(std::ceil(val) == val)
        {
            new_dt = 2.0 * old_dt;
        }
        else
        {
            new_dt = old_dt;
        }
    }
    else
    {
        //std::cerr << "this will never happen...I promise" << std::endl;
        new_dt = old_dt;
    }

    //if (new_dt <= D_TIME_MIN)
    if (new_dt < D_TIME_MIN)
    {
        new_dt = D_TIME_MIN;
    }
    //else if (new_dt >= D_TIME_MAX)
    else if (new_dt > D_TIME_MAX)
    {
        new_dt = D_TIME_MAX;
    }

    return new_dt;
}
