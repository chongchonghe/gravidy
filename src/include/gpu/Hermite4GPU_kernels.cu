#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016
 *
 * Cristián Maureira-Fredes <cmaureirafredes@gmail.com>
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright
 * notice, this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright
 * notice, this list of conditions and the following disclaimer in the
 * documentation and/or other materials provided with the distribution.
 *
 * 3. The name of the author may not be used to endorse or promote
 * products derived from this software without specific prior written
 * permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR ``AS IS'' AND ANY EXPRESS
 * OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE
 * GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER
 * IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
 * OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN
 * IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 */
#undef _GLIBCXX_ATOMIC_BUILTINS
#include "Hermite4GPU.cuh"

/*
 * @fn k_init_acc_jr
 *
 *
 * @desc GPU Kernel which calculates the initial acceleration and jerk
 * of all the particles of the system.
 *
 */
__global__ void k_init_acc_jrk (Predictor *p,
                                Forces *f,
                                int n,
                                double e2,
                                int dev,
                                int dev_size)
{

    extern __shared__ Predictor sh[];

    Forces ff;
    ff.a[0]  = 0.0;
    ff.a[1]  = 0.0;
    ff.a[2]  = 0.0;
    ff.a1[0] = 0.0;
    ff.a1[1] = 0.0;
    ff.a1[2] = 0.0;

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int tx = threadIdx.x;

    if (id < dev_size)
    {
      Predictor pred = p[id+(dev*dev_size)];
      //Predictor pred = p[id];
      int tile = 0;
      for (int i = 0; i < n; i += BSIZE)
      {
          int idx = tile * BSIZE + tx;
          sh[tx]   = p[idx];
          __syncthreads();
          for (int k = 0; k < BSIZE; k++)
          {
              k_force_calculation(pred, sh[k], ff, e2);
          }
          __syncthreads();
          tile++;
      }
      f[id] = ff;
    }
}

__device__ void k_force_calculation(const Predictor &i_p,
                                    const Predictor &j_p,
                                    Forces &f,
                                    const double &e2)
{
    double rx = j_p.r[0] - i_p.r[0];
    double ry = j_p.r[1] - i_p.r[1];
    double rz = j_p.r[2] - i_p.r[2];

    double vx = j_p.v[0] - i_p.v[0];
    double vy = j_p.v[1] - i_p.v[1];
    double vz = j_p.v[2] - i_p.v[2];

    double r2     = rx*rx + ry*ry + rz*rz + e2;
    double rinv   = rsqrt(r2);
    double r2inv  = rinv  * rinv;
    double r3inv  = r2inv * rinv;
    double r5inv  = r2inv * r3inv;
    double mr3inv = r3inv * j_p.m;
    double mr5inv = r5inv * j_p.m;

    double rv = rx*vx + ry*vy + rz*vz;

    f.a[0] += (rx * mr3inv);
    f.a[1] += (ry * mr3inv);
    f.a[2] += (rz * mr3inv);

    f.a1[0] += (vx * mr3inv - (3 * rv) * rx * mr5inv);
    f.a1[1] += (vy * mr3inv - (3 * rv) * ry * mr5inv);
    f.a1[2] += (vz * mr3inv - (3 * rv) * rz * mr5inv);
}
/*
 * @fn k_prediction
 *
 *
 * @desc GPU Kernel which calculates the predictors
 *
 */
__global__ void k_prediction(Forces *f,
                             double4 *r,
                             double4 *v,
                             double *t,
                             Predictor *p,
                             int dev_size,
                             double ITIME)
{

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < dev_size)
    {
        double dt  = ITIME - t[i];
        double dt2 = 0.5 * (dt * dt);
        double dt3 = 0.166666666666666 * (dt * dt * dt);

        Forces ff = f[i];
        double4 rr = r[i];
        double4 vv = v[i];

        p[i].r[0] = (dt3 * ff.a1[0]) + (dt2 * ff.a[0]) + (dt * vv.x) + rr.x;
        p[i].r[1] = (dt3 * ff.a1[1]) + (dt2 * ff.a[1]) + (dt * vv.y) + rr.y;
        p[i].r[2] = (dt3 * ff.a1[2]) + (dt2 * ff.a[2]) + (dt * vv.z) + rr.z;

        p[i].v[0] = (dt2 * ff.a1[0]) + (dt * ff.a[0]) + vv.x;
        p[i].v[1] = (dt2 * ff.a1[1]) + (dt * ff.a[1]) + vv.y;
        p[i].v[2] = (dt2 * ff.a1[2]) + (dt * ff.a[2]) + vv.z;

        p[i].m = rr.w;
    }
}

/*
 * @fn k_update()
 *
 * @brief Gravitational interaction kernel.
 */
__global__ void k_update(Predictor *i_p,
                         Predictor *j_p,
                         Forces *fout,
                         int n,
                         int total,
                         double e2)
{
    int ibid = blockIdx.x;
    int jbid = blockIdx.y;
    int tid  = threadIdx.x;
    int iaddr  = tid + blockDim.x * ibid;
    int jstart = (n * (jbid  )) / NJBLOCK;
    int jend   = (n * (jbid+1)) / NJBLOCK;

    Predictor ip = i_p[iaddr];
    Forces fo;
    fo.a[0] = 0.0;
    fo.a[1] = 0.0;
    fo.a[2] = 0.0;
    fo.a1[0] = 0.0;
    fo.a1[1] = 0.0;
    fo.a1[2] = 0.0;

        for(int j=jstart; j<jend; j+=BSIZE)
        {
            __shared__ Predictor jpshare[BSIZE];
            __syncthreads();
            Predictor *src = (Predictor *)&j_p[j];
            Predictor *dst = (Predictor *)jpshare;
            dst[      tid] = src[      tid];
            dst[BSIZE+tid] = src[BSIZE+tid];
            __syncthreads();

            // If the total amount of particles is not a multiple of BSIZE
            if(jend-j < BSIZE)
            {
                #pragma unroll 4
                for(int jj=0; jj<jend-j; jj++)
                {
                    Predictor jp = jpshare[jj];
                    k_force_calculation(ip, jp, fo, e2);
                }
            }
            else
            {
                #pragma unroll 4
                for(int jj=0; jj<BSIZE; jj++)
                {
                    Predictor jp = jpshare[jj];
                    k_force_calculation(ip, jp, fo, e2);
                }
            }
        }
        fout[iaddr*NJBLOCK + jbid] = fo;
}

/*
 * @fn k_reduce()
 *
 * @brief Forces reduction kernel
 */
__global__ void k_reduce(Forces *in,
                       Forces *out,
                       int shift_id,
                       int shift)
{
    extern __shared__ Forces sdata[];

    const int xid   = threadIdx.x;
    const int bid   = blockIdx.x;
    const int iaddr = xid + blockDim.x * bid;

    sdata[xid] = in[iaddr+shift*NJBLOCK];
    __syncthreads();

    if(xid < 8) sdata[xid] += sdata[xid + 8];
    if(xid < 4) sdata[xid] += sdata[xid + 4];
    if(xid < 2) sdata[xid] += sdata[xid + 2];
    if(xid < 1) sdata[xid] += sdata[xid + 1];

    if(xid == 0){
        out[bid] = sdata[0];
    }
}

__global__ void k_energy(double4 *r,
                         double4 *v,
                         double *ekin,
                         double *epot,
                         int n,
                         int dev_size,
                         int dev)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j;
    double ekin_tmp = 0.0;
    int id = i+dev*dev_size;

    if (i < dev_size)
    {
        double epot_tmp = 0.0;
        double4 ri = r[id];
        double4 vi = v[id];
        for (j = id+1; j < n; j++)
        {
            double rx = r[j].x - ri.x;
            double ry = r[j].y - ri.y;
            double rz = r[j].z - ri.z;
            double r2 = rx*rx + ry*ry + rz*rz;

            epot_tmp -= (ri.w * r[j].w) * rsqrt(r2);
        }

        double vx = vi.x * vi.x;
        double vy = vi.y * vi.y;
        double vz = vi.z * vi.z;
        double v2 = vx + vy + vz;

        ekin_tmp = 0.5 * ri.w * v2;

        ekin[i] = ekin_tmp;
        epot[i] = epot_tmp;
    }
}





__global__ void k_correction(unsigned int *move,
                             Forces *f,
                             Forces *old,
                             Predictor *p,
                             double4 *r,
                             double4 *v,
                             double *t,
                             double *dt,
                             double3 *a2,
                             double3 *a3,
                             unsigned int dev_size,
                             double ITIME,
                             double ETA)
{
  // thread index
  int thread_idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (thread_idx < dev_size)
  {
      // i is the particle to move (gets taken from (ns->h_)move)
      // i is an index into all the other arrays
      int i = move[thread_idx];

      Forces ff = f[i];
      Forces oo = old[i];
      Predictor pp = p[i];

      double dt1 = dt[i];
      double dt2 = dt1 * dt1;
      double dt3 = dt2 * dt1;
      double dt4 = dt2 * dt2;
      double dt5 = dt4 * dt1;

      double dt2inv = 1.0/dt2;
      double dt3inv = 1.0/dt3;

      double dt3_6 = 0.166666666666666*dt3;
      double dt4_24 = 0.041666666666666*dt4;
      double dt5_120 = 0.008333333333333*dt5;

      // Acceleration 2nd derivate
      a2[i].x = (-6 * (oo.a[0] - ff.a[0] ) - dt1 * (4 * oo.a1[0] + 2 * ff.a1[0]) ) * dt2inv;
      a2[i].y = (-6 * (oo.a[1] - ff.a[1] ) - dt1 * (4 * oo.a1[1] + 2 * ff.a1[1]) ) * dt2inv;
      a2[i].z = (-6 * (oo.a[2] - ff.a[2] ) - dt1 * (4 * oo.a1[2] + 2 * ff.a1[2]) ) * dt2inv;

      // Acceleration 3rd derivate
      a3[i].x = (12 * (oo.a[0] - ff.a[0] ) + 6 * dt1 * (oo.a1[0] + ff.a1[0]) ) * dt3inv;
      a3[i].y = (12 * (oo.a[1] - ff.a[1] ) + 6 * dt1 * (oo.a1[1] + ff.a1[1]) ) * dt3inv;
      a3[i].z = (12 * (oo.a[2] - ff.a[2] ) + 6 * dt1 * (oo.a1[2] + ff.a1[2]) ) * dt3inv;


      // Correcting position
      r[i].x = pp.r[0] + (dt4_24)*a2[i].x + (dt5_120)*a3[i].x;
      r[i].y = pp.r[1] + (dt4_24)*a2[i].y + (dt5_120)*a3[i].y;
      r[i].z = pp.r[2] + (dt4_24)*a2[i].z + (dt5_120)*a3[i].z;

      // Correcting velocity
      v[i].x = pp.v[0] + (dt3_6)*a2[i].x +   (dt4_24)*a3[i].x;
      v[i].y = pp.v[1] + (dt3_6)*a2[i].y +   (dt4_24)*a3[i].y;
      v[i].z = pp.v[2] + (dt3_6)*a2[i].z +   (dt4_24)*a3[i].z;

      t[i] = ITIME;
      double normal_dt = k_get_timestep_normal(ETA, a2[i], a3[i], dt[i], ff);
      dt[i] = k_normalize_dt(normal_dt, dt[i], t[i]);

  }

}

/** Vector magnitude calculation; copied from the one in NbodyUtils **/
__device__ double k_get_magnitude(const double &x, const double &y, const double &z)
{
  return sqrt(x*x + y*y + z*z);
}

/** Time step calculation; copied from the one in NbodyUtils.
Used to take an unsigned int i argument but I got rid if it.
**/
__device__ double k_get_timestep_normal(const float &ETA,
                                 const double3 &a2,
                                 const double3 &a3,
                                 const double &dt,
                                 const Forces &f)
{
  // Calculating a_{1,i}^{(2)} = a_{0,i}^{(2)} + dt * a_{0,i}^{(3)}
  double ax1_2 = a2.x + dt * a3.x;
  double ay1_2 = a2.y + dt * a3.y;
  double az1_2 = a2.z + dt * a3.z;

  // |a_{1,i}|
  double abs_a1 = k_get_magnitude(f.a[0],
                                f.a[1],
                                f.a[2]);
  // |j_{1,i}|
  double abs_j1 = k_get_magnitude(f.a1[0],
                                f.a1[1],
                                f.a1[2]);
  // |j_{1,i}|^{2}
  double abs_j12  = abs_j1 * abs_j1;
  // a_{1,i}^{(3)} = a_{0,i}^{(3)} because the 3rd-order interpolation
  double abs_a1_3 = k_get_magnitude(a3.x,
                                  a3.y,
                                  a3.z);
  // |a_{1,i}^{(2)}|
  double abs_a1_2 = k_get_magnitude(ax1_2, ay1_2, az1_2);
  // |a_{1,i}^{(2)}|^{2}
  double abs_a1_22  = abs_a1_2 * abs_a1_2;

  // variable used to be called "normal_dt" and was returned (just skipping the new variable declaration)
  return sqrt(ETA * ((abs_a1 * abs_a1_2 + abs_j12) / (abs_j1 * abs_a1_3 + abs_a1_22)));
}

/** Normalization of the timestep.
 * This method take care of the limits conditions to avoid large jumps between
 * the timestep distribution
 Copied from the version in NbodyUtils; that version takes an argument "unsigned int i"
 but does not use it, so I dropped that argument.

 For this, the local copy of new_dt is rewritten a lot, so I'm letting that one
 be non-constant. In k_correction, a newly declared variable (not used after this)
 is passed in as new_dt, so I will let it be pass-by-reference since we don't need
 to keep that data safe past this function.
 old_dt and t are still pass by reference and constant.
 */
__device__ double k_normalize_dt(double &new_dt,
                          const double &old_dt,
                          const double &t)
{
  if (new_dt <= old_dt/8)
  {
      new_dt = D_TIME_MIN;
  }
  else if ( old_dt/8 < new_dt && new_dt <= old_dt/4)
  {
      new_dt = old_dt / 8;
  }
  else if ( old_dt/4 < new_dt && new_dt <= old_dt/2)
  {
      new_dt = old_dt / 4;
  }
  else if ( old_dt/2 < new_dt && new_dt <= old_dt)
  {
      new_dt = old_dt / 2;
  }
  else if ( old_dt < new_dt && new_dt <= old_dt * 2)
  {
      new_dt = old_dt;
  }
  else if (2 * old_dt < new_dt)
  {
      double val = t/(2 * old_dt);
      //float val = t/(2 * old_dt);
      if(std::ceil(val) == val)
      {
          new_dt = 2.0 * old_dt;
      }
      else
      {
          new_dt = old_dt;
      }
  }
  else
  {
      //std::cerr << "this will never happen...I promise" << std::endl;
      new_dt = old_dt;
  }

  //if (new_dt <= D_TIME_MIN)
  if (new_dt < D_TIME_MIN)
  {
      new_dt = D_TIME_MIN;
  }
  //else if (new_dt >= D_TIME_MAX)
  else if (new_dt > D_TIME_MAX)
  {
      new_dt = D_TIME_MAX;
  }

  return new_dt;
}
